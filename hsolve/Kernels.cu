#include "hip/hip_runtime.h"
#include "HSolveActive.h"

__global__
void get_lookup_rows_and_fractions_cuda(
		double* lookups,
		double* table,
		double min, double max, double dx,
		int* rows, double* fracs,
		unsigned int nColumns, unsigned int size){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < size){
		double x = lookups[tid];

		if ( x < min )
			x = min;
		else if ( x > max )
			x = max;

		double div = ( x - min ) / dx;
		unsigned int integer = ( unsigned int )( div );

		rows[tid] = integer*nColumns;
		fracs[tid] = div-integer;
	}
}

void HSolveActive::get_lookup_rows_and_fractions_cuda_wrapper(int gpu_load_count){
	int num_comps = V_.size();

	int THREADS_PER_BLOCK = 512;
	int BLOCKS = gpu_load_count/THREADS_PER_BLOCK;
	BLOCKS = (gpu_load_count + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;


	// Getting lookup metadata for Vm
	get_lookup_rows_and_fractions_cuda<<<BLOCKS,THREADS_PER_BLOCK>>>(u_V,
    		d_V_table,
    		vTable_.get_min(), vTable_.get_max(), vTable_.get_dx(),
    		u_V_rows, u_V_fracs,
    		vTable_.get_num_of_columns(), gpu_load_count);
}

/*
 * Based on the near lookup value and fraction value, the function
 * interpolates the value and uses it to update appropriate state variables.
 * "indices" array is a subset of compartment id's which are
 * voltage dependent gate indices or Calcium dependent gate indices
 */
__global__
void advance_channels_opt_cuda(
		int* rows,
		double* fracs,
		double* table,
		int* indices,
		int* gate_to_comp,
		double* gate_values,
		int* gate_columns,
		int* state2chanId,
		int* chan_instants,
		unsigned int nColumns,
		double dt,
		int size
		){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid < size){
		double a,b,C1,C2;
		int index, lookup_index, row_start_index, column;

		index = indices[tid];
		lookup_index = gate_to_comp[tid];
		row_start_index = rows[lookup_index];
		column = gate_columns[index];

		a = table[row_start_index + column];
		b = table[row_start_index + column + nColumns];

		C1 = a + (b-a)*fracs[lookup_index];

		a = table[row_start_index + column + 1];
		b = table[row_start_index + column + 1 + nColumns];

		C2 = a + (b-a)*fracs[lookup_index];

		if(!chan_instants[state2chanId[tid]]){
			a = 1.0 + dt/2.0 * C2; // reusing a
			gate_values[index] = ( gate_values[index] * ( 2.0 - a ) + dt * C1 ) / a;
		}
		else{
			gate_values[index] = C1/C2;
		}
	}
}

/*
 * Advance Channels performing both lookup and state update
 */
__global__
void advance_channels_for_externalCalcium(
		double* d_externalCalcium,
		int* d_exCalgate_indices,
		int* state2chanId,
		double* gate_values,
		int* state2Column,
		int* chan_instants,
		double* table,
		double min, double max, double dx,
		unsigned int nColumns, double dt, unsigned int size){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < size){
		int index = d_exCalgate_indices[tid]; // Index in the state_ array
		int chan_id = state2chanId[index];
		// Update state only if there is a contribution
		if(d_externalCalcium[chan_id] != 0){
			double a,b,C1,C2;
			double x = d_externalCalcium[chan_id];

			if ( x < min )
				x = min;
			else if ( x > max )
				x = max;

			double div = ( x - min ) / dx;
			unsigned int integer = ( unsigned int )( div );

			int row_start_index = integer*nColumns;
			double frac = div-integer;

			// Perform the update
			int column = state2Column[index];

			a = table[row_start_index + column];
			b = table[row_start_index + column + nColumns];

			C1 = a + (b-a)*frac;

			a = table[row_start_index + column + 1];
			b = table[row_start_index + column + 1 + nColumns];

			C2 = a + (b-a)*frac;

			if(!chan_instants[chan_id]){
				a = 1.0 + dt/2.0 * C2; // reusing a
				gate_values[index] = ( gate_values[index] * ( 2.0 - a ) + dt * C1 ) / a;
			}
			else{
				gate_values[index] = C1/C2;
			}

		}
	}
}

